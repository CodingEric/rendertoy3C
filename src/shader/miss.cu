#include "hip/hip_runtime.h"
#include <src/shader/shader_common.h>

static __forceinline__ __device__ rendertoy3o::RadiancePRD loadMissRadiancePRD()
{
    rendertoy3o::RadiancePRD prd = {};
    return prd;
}

static __forceinline__ __device__ void storeMissRadiancePRD( rendertoy3o::RadiancePRD prd )
{
    optixSetPayload_5( __float_as_uint( prd.emitted.x ) );
    optixSetPayload_6( __float_as_uint( prd.emitted.y ) );
    optixSetPayload_7( __float_as_uint( prd.emitted.z ) );

    optixSetPayload_8( __float_as_uint( prd.radiance.x ) );
    optixSetPayload_9( __float_as_uint( prd.radiance.y ) );
    optixSetPayload_10( __float_as_uint( prd.radiance.z ) );

    optixSetPayload_17( prd.done );
}

extern "C" __global__ void __miss__radiance()
{
    optixSetPayloadTypes( rendertoy3o::PAYLOAD_TYPE_RADIANCE );

    rendertoy3o::MissData* rt_data  = reinterpret_cast<rendertoy3o::MissData*>( optixGetSbtDataPointer() );
    rendertoy3o::RadiancePRD prd = loadMissRadiancePRD();

    // prd.radiance  = make_float3( rt_data->bg_color );
    prd.radiance = optixDirectCall<float3>(0);
    prd.emitted   = make_float3( 0.f );
    prd.done      = true;

    storeMissRadiancePRD( prd );
}