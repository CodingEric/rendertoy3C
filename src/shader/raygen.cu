#include "hip/hip_runtime.h"
#include <src/shader/shader_common.h>

extern "C"
{
    __constant__ rendertoy3o::RenderSettings params;
}

//------------------------------------------------------------------------------
//
// Programs
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
    const int w = params.film_settings.width;
    const int h = params.film_settings.height;
    const float3 eye = params.camera_settings.eye;
    const float3 U = params.camera_settings.U;
    const float3 V = params.camera_settings.V;
    const float3 W = params.camera_settings.W;
    const uint3 idx = optixGetLaunchIndex();
    const int subframe_index = params.film_settings.subframe_index;

    unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

    float3 result = make_float3(0.0f);
    int i = params.film_settings.samples_per_launch;
    do
    {
        // The center of each pixel is at fraction (0.5,0.5)
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

        const float2 d = 2.0f * make_float2(
                                    (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
                                    (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)) -
                         1.0f;
        float3 ray_direction = normalize(d.x * U + d.y * V + W);
        float3 ray_origin = eye;

        rendertoy3o::RadiancePRD prd;
        prd.attenuation = make_float3(1.f);
        prd.seed = seed;
        prd.depth = 0;

        float3 last_attenuation = prd.attenuation;

        for (;;)
        {
            rendertoy3o::traceRadiance(
                params.handle,
                ray_origin,
                ray_direction,
                0.01f, // tmin       // TODO: smarter offset
                1e16f, // tmax
                prd);

            result += prd.emitted;
            result += prd.radiance * last_attenuation;
            last_attenuation = prd.attenuation;

            const float p = dot(prd.attenuation, make_float3(0.30f, 0.59f, 0.11f));
            const bool done = prd.done || rnd(prd.seed) > p;
            if (done)
                break;
            prd.attenuation /= p;

            ray_origin = prd.origin;
            ray_direction = prd.direction;

            ++prd.depth;
        }
    } while (--i);

    const uint3 launch_index = optixGetLaunchIndex();
    const unsigned int image_index = launch_index.y * params.film_settings.width + launch_index.x;
    float3 accum_color = result / static_cast<float>(params.film_settings.samples_per_launch);

    if (subframe_index > 0)
    {
        const float a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.film_settings.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.film_settings.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
    params.film_settings.frame_buffer[image_index] = make_color(accum_color);
}
